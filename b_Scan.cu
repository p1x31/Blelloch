#include "hip/hip_runtime.h"
/*
    ID: 1566767
	Name: Vadim Makarenkov


	Goals implemented:
		- Block scan
		- Full scan for arbitrary length large vectors
			one block scan of 3-level block scan depending on the size of the vector
		- BCAO for both scans

	Hardware:
		CPU - Intel Core i5-6600k @ 3.5GHz
		GPU - NVIDIA GeForce GTX 960/970

	Timings:
		10,000,000 Elements
		  host     : 8.68400ms
		  gpu      : 4.23866ms
		  gpu bcao : 3.42429ms

	Additional work:

*/

#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <helper_functions.h>
#include <math.h>
#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#define BLOCK_SIZE 128
#define BLOCK_SIZE_TWICE BLOCK_SIZE*2

// Avoiding Bank Conflicts
#define NUM_BANKS 32
#define LOG_NUM_BANKS 5
// There were two BCAO optimisations in the paper - this one is fastest
#define CONFLICT_FREE_OFFSET(n)  (((n) >> LOG_NUM_BANKS) + ((n) >> (2 * LOG_NUM_BANKS)))

#define CUDA_ERROR( err, msg ) { \
if (err != hipSuccess) {\
    printf( "%s: %s in %s at line %d\n", msg, hipGetErrorString(err), __FILE__, __LINE__);\
    exit( EXIT_FAILURE );\
}\
}

//#define CUDA_ERROR( message, err) {\
//	if (err != hipSuccess) { \
//		printf(stderr, message); \
//		fprintf(stderr, ": %s\n", hipGetErrorString(err));\
//		exit(0);\
//	}\
//}
// sequential scan version for speed and correctness comparison
// sequential scan on CPU
__host__
void sequential_scan(int *g_idata, int *g_odata, int n) {

	g_odata[0] = 0;
	for (int i = 1; i < n; i++) {
		g_odata[i] = g_odata[i - 1] + g_idata[i - 1];
	}
}


// for comparing the results with the host version
static void compare_results(const int *output, const int *input,
		int numElements) {
	for (int i = 0; i < numElements; ++i) {
		if (fabs(output[i] != input[i]) ) {
			printf("%d ----------- %d\n", output[i], input[i]);
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}
	}
}


// multilevel block scan helper
__global__
void add_to_block(int *block, int len_block, int *SUM) {
	// get the value of the element that has to be
	// added to the main vector. We do not need to worry
	// about accessing out of bounds because this function
	// is called with the same number of blocks as the size of SUM array.
	int s = SUM[blockIdx.x];

	// get the address the vector that has to be updated
	// Skip loads and stores for inactive threads of last threadblock (pos >= N)
	int pos = blockIdx.x * BLOCK_SIZE_TWICE + threadIdx.x;

	__syncthreads();
	// update two elements in the vector
	if (pos < len_block)
		block[pos] += s;
	if (pos + blockDim.x < len_block)
		block[pos + blockDim.x] += s;
}

__global__
void block_scan_full_BCAO(int *g_idata, int *g_odata, int n, int *SUM,
		int add_last) {

	// shared memory initialised to contain more than
	// twice memory due to the offset because it might
	// Avoid out-of-bound access
	__shared__ int temp[BLOCK_SIZE_TWICE + (BLOCK_SIZE >> 4)];

	// local variables for the later usage to improve the performance
	// https://developer.nvidia.com/gpugems/GPUGems3/gpugems3_ch39.html
	int thid = threadIdx.x;
	int thid_shift = thid << 1;
	int blockId = blockIdx.x * (BLOCK_SIZE << 1);
	int blockOffset= 0;
	int last = 0;


	// offset to avoid bank conflicts
	int ai = thid;
	int bi = thid + BLOCK_SIZE;
	int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
	int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

	// load the elements from global memory into the shared memory
	if (blockId + ai < n)
		temp[ai + bankOffsetA] = g_idata[blockId + ai];
	else
		temp[ai + bankOffsetA] = 0;
	if (blockId + bi < n)
		temp[bi + bankOffsetB] = g_idata[blockId + bi];
	else
		temp[bi + bankOffsetB] = 0;

	// save the last element for later to improve the performance
	if (add_last && thid == BLOCK_SIZE - 1)
		last = temp[BLOCK_SIZE_TWICE - 1
				+ CONFLICT_FREE_OFFSET((BLOCK_SIZE << 1) - 1)];

	// https://developer.nvidia.com/gpugems/GPUGems3/gpugems3_ch39.html
	// build sum in place up the tree (reduction phase)
	for (int d = BLOCK_SIZE; d > 0; d >>= 1) {
		__syncthreads();
		if (thid < d) {
			int ai = ((thid_shift + 1) << blockOffset) - 1;
			int bi = ((thid_shift + 2) << blockOffset) - 1;

			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
			temp[bi] += temp[ai];
		}
		blockOffset++;
	}

	// update the last element to 0
	if (thid == 0) {
		temp[BLOCK_SIZE_TWICE - 1 + CONFLICT_FREE_OFFSET(BLOCK_SIZE_TWICE - 1)] =
				0;
	}

	// traverse down tree & build scan (distribution phase)
	for (int d = 1; d < BLOCK_SIZE_TWICE; d <<= 1) {
		blockOffset--;
		__syncthreads();
		if (thid < d) {
			int ai = ((thid_shift + 1) << blockOffset) - 1;
			int bi = ((thid_shift + 2) << blockOffset) - 1;
			// https://developer.nvidia.com/gpugems/GPUGems3/gpugems3_ch39.html
			ai += CONFLICT_FREE_OFFSET(ai);
			bi += CONFLICT_FREE_OFFSET(bi);
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}

	__syncthreads();
	// extract the sum (merged to improve the performance)
	if (add_last && thid == BLOCK_SIZE - 1)
		SUM[blockIdx.x] = temp[BLOCK_SIZE_TWICE - 1
				+ CONFLICT_FREE_OFFSET(BLOCK_SIZE_TWICE - 1)] + last;

	// update the output vector by loading shared memory into the global memory
	if (blockId + ai < n)
		g_odata[blockId + ai] = temp[ai + bankOffsetA];
	if (blockId + bi < n)
		g_odata[blockId + bi] = temp[bi + bankOffsetB];
}

__host__
void full_block_scan_BCAO(int *h_IN, int *h_OUT, int len) {

	// Initialisation

	// error code to check return values for CUDA class
	hipError_t err = hipSuccess;

	// size to allocate for the vectors
	size_t size = len * sizeof(int);

	// create device timer
	hipEvent_t d_start, d_stop;
	float d_msecs;
	hipEventCreate(&d_start);
	hipEventCreate(&d_stop);

	// allocate memory for all the possible vectors needed for the execution
	int *d_IN = NULL;
	err = hipMalloc((void **) &d_IN, size);
	CUDA_ERROR(err, "Failed to allocate device vector IN");

	int *d_OUT = NULL;
	err = hipMalloc((void**) &d_OUT, size);
	CUDA_ERROR(err, "Failed to allocate device vector OUT");

	int *d_SUM_1 = NULL;
	err = hipMalloc((void**) &d_SUM_1,
			(1 + ((len - 1) / (BLOCK_SIZE * 2))) * sizeof(int));


	int *d_SUM_1_Scanned = NULL;
	err = hipMalloc((void**) &d_SUM_1_Scanned,
			(1 + ((len - 1) / (BLOCK_SIZE * 2))) * sizeof(int));


	int *d_SUM_2 = NULL;
	err = hipMalloc((void**) &d_SUM_2, (BLOCK_SIZE << 1) * sizeof(int));


	int *d_SUM_2_Scanned = NULL;
	err = hipMalloc((void**) &d_SUM_2_Scanned,
			(BLOCK_SIZE << 1) * sizeof(int));


	// copy the memory from the host to the device
	err = hipMemcpy(d_IN, h_IN, size, hipMemcpyHostToDevice);


	// size of the grid for each level
	int blocksPerGridLevel1 = 1 + ((len - 1) / (BLOCK_SIZE * 2));
	int blocksPerGridLevel2 = 1 + ceil(blocksPerGridLevel1 / (BLOCK_SIZE << 1));
	int blocksPerGridLevel3 = 1 + ceil(blocksPerGridLevel2 / (BLOCK_SIZE << 1));


	// sumsArrThreadsNeeded > THREADS_PER_BLOCK
	// choosing the level on which to run the kernels
	// based on the size of the grids

	// if level one grid size is equal to 1 then single
	// LEVEL 1 is enough to scan the whole array
	if (blocksPerGridLevel1 == 1) {
		// record the start time
		hipEventRecord(d_start, 0);
		// execute the actual kernel
		block_scan_full_BCAO<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_IN, d_OUT,
				len,
				NULL, 0);

		// record the stop time
		hipEventRecord(d_stop, 0);
		hipEventSynchronize(d_stop);
		hipDeviceSynchronize();
	}

	// if level two grid size is equal to 1 then two (LEVEL 2)
	// scans are required to scan the whole array
	else if (blocksPerGridLevel2 == 1) {
		// record the start time
		hipEventRecord(d_start, 0);

		// execute the actual kernels
		block_scan_full_BCAO<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_IN, d_OUT,
				len, d_SUM_1, 1);
		block_scan_full_BCAO<<<blocksPerGridLevel2, BLOCK_SIZE>>>(d_SUM_1,
				d_SUM_1_Scanned, blocksPerGridLevel1, NULL, 0);
		add_to_block<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_OUT, len,
				d_SUM_1_Scanned);

		// record the stop time
		hipEventRecord(d_stop, 0);
		hipEventSynchronize(d_stop);
		hipDeviceSynchronize();
	}

	//Final step of large-array scan: combine basic inclusive scan with exclusive scan of top elements of input arrays
	else if (blocksPerGridLevel3 == 1) {
		// record the start time
		hipEventRecord(d_start, 0);

		// execute the actual kernels
		block_scan_full_BCAO<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_IN, d_OUT,
				len, d_SUM_1, 1);
		block_scan_full_BCAO<<<blocksPerGridLevel2, BLOCK_SIZE>>>(d_SUM_1,
				d_SUM_1_Scanned, blocksPerGridLevel1, d_SUM_2, 1);
		block_scan_full_BCAO<<<1, BLOCK_SIZE>>>(d_SUM_2, d_SUM_2_Scanned,
				blocksPerGridLevel2, NULL, 0);
		add_to_block<<<blocksPerGridLevel2, BLOCK_SIZE>>>(d_SUM_1_Scanned,
				blocksPerGridLevel1, d_SUM_2_Scanned);
		add_to_block<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_OUT, len,
				d_SUM_1_Scanned);

		// record the stop time
		hipEventRecord(d_stop, 0);
		hipEventSynchronize(d_stop);
		hipDeviceSynchronize();
	}

	// if none of the conditions above is met, it means that the array is too
	// large to be scanned in 3 level scan, therefore we print the error message
	// and return
	else {
		fprintf(stderr,
				"The array size=%d is too large to be scanned with level 3 scan!\n",
				len);

		// using goto is discouraged, however, in such situations
		// where in the error conditions exit or cleanup is required
		goto cleanup;
	}

	// check whether the run was successful
	err = hipGetLastError();


	// get the duration it took for the kernels to execute
	err = hipEventElapsedTime(&d_msecs, d_start, d_stop);


	// print the time elapsed
	printf(
			"Full block with bank avoidance scan with %d elements took = %.5fmSecs\n",
			len, d_msecs);

	// copy the result from the device back to the host
	err = hipMemcpy(h_OUT, d_OUT, size, hipMemcpyDeviceToHost);

	cleanup:
		// Free device global memory
		CUDA_ERROR(hipFree(d_IN), "Failed to free device vector IN");
		CUDA_ERROR(hipFree(d_OUT), "Failed to free device vector OUT");
		CUDA_ERROR(hipFree(d_SUM_1), "Failed to free device vector SUM_1");
		CUDA_ERROR(hipFree(d_SUM_1_Scanned),
				"Failed to free device vector SUM_1_Scanned");
		CUDA_ERROR(hipFree(d_SUM_2), "Failed to free device vector SUM_2");
		CUDA_ERROR(hipFree(d_SUM_2_Scanned),
				"Failed to free device vector SUM_2_Scanned");

		// Clean up the Device timer event objects
		hipEventDestroy(d_start);
		hipEventDestroy(d_stop);

		// Reset the device and exit
		err = hipDeviceReset();
		CUDA_ERROR(err, "Failed to reset the device");


}

__global__
void block_scan_full(int *g_idata, int *g_odata, int n, int *SUM,
		int add_last) {
	// shared memory init
	__shared__ int temp[BLOCK_SIZE << 1];

	// local variables for the later usage to improve the performance
	int thid = threadIdx.x;
	int blockId = blockDim.x * blockIdx.x << 1;
	int offset = 0;
	int last = 0;

	// load the elements from global memory into the shared memory
	if (blockId + (thid << 1) < n)
		temp[thid << 1] = g_idata[blockId + (thid << 1)];
	if (blockId + (thid << 1) + 1 < n)
		temp[(thid << 1) + 1] = g_idata[blockId + (thid << 1) + 1];

	// save the last element for later to improve the performance
	if (add_last && thid == BLOCK_SIZE - 1)
		last = temp[(thid << 1) + 1];

	// build sum in place up the tree (reduction phase)
	for (int d = BLOCK_SIZE; d > 0; d >>= 1) {
		__syncthreads();
		if (thid < d) {
			int ai = (((thid << 1) + 1) << offset) - 1;
			int bi = (((thid << 1) + 2) << offset) - 1;
			temp[bi] += temp[ai];
		}
		offset++;
	}

	// clear the last element
	if (thid == 0)
		temp[(BLOCK_SIZE << 1) - 1] = 0;

	// traverse down tree & build scan (distribution phase)
	for (int d = 1; d < (BLOCK_SIZE << 1); d <<= 1) {
		offset--;
		__syncthreads();
		if (thid < d) {
			int ai = (((thid << 1) + 1) << offset) - 1;
			int bi = (((thid << 1) + 2) << offset) - 1;
			int t = temp[ai];
			temp[ai] = temp[bi];
			temp[bi] += t;
		}
	}

	__syncthreads();
	// extract the sum (merged to improve the performance)
	if (add_last && thid == BLOCK_SIZE - 1)
		SUM[blockIdx.x] = temp[(thid << 1) + 1] + last;

	// update the output vector by loading shared memory into the global memory
	if (blockId + (thid << 1) < n)
		g_odata[blockId + (thid << 1)] = temp[thid << 1];
	if (blockId + (thid << 1) + 1 < n)
		g_odata[blockId + (thid << 1) + 1] = temp[(thid << 1) + 1];
}

__host__
void full_block_scan(int *h_IN, int *h_OUT, int len) {

	// error code to check return values for CUDA class
	hipError_t err = hipSuccess;

	// size to allocate for the vectors
	size_t size = len * sizeof(int);

	// create device timer
	hipEvent_t d_start, d_stop;
	float d_msecs;
	hipEventCreate(&d_start);
	hipEventCreate(&d_stop);

	// allocate memory for all the possible vectors needed for the execution
	int *d_IN = NULL;
	err = hipMalloc((void **) &d_IN, size);
	CUDA_ERROR(err, "Failed to allocate device vector IN");

	int *d_OUT = NULL;
	err = hipMalloc((void**) &d_OUT, size);
	CUDA_ERROR(err, "Failed to allocate device vector OUT");

	int *d_SUM_1 = NULL;
	err = hipMalloc((void**) &d_SUM_1,
			(1 + ((len - 1) / (BLOCK_SIZE * 2))) * sizeof(int));
	CUDA_ERROR(err ,"Failed to allocate device vector OUT");

	int *d_SUM_1_Scanned = NULL;
	err = hipMalloc((void**) &d_SUM_1_Scanned,
			(1 + ((len - 1) / (BLOCK_SIZE * 2))) * sizeof(int));


	int *d_SUM_2 = NULL;
	err = hipMalloc((void**) &d_SUM_2, (BLOCK_SIZE << 1) * sizeof(int));


	int *d_SUM_2_Scanned = NULL;
	err = hipMalloc((void**) &d_SUM_2_Scanned,
			(BLOCK_SIZE << 1) * sizeof(int));


	// copy the memory from the host to the device
	err = hipMemcpy(d_IN, h_IN, size, hipMemcpyHostToDevice);


	// size of the grid for each level
	int blocksPerGridLevel1 = 1 + ((len - 1) / (BLOCK_SIZE * 2));
	int blocksPerGridLevel2 = 1 + ceil(blocksPerGridLevel1 / (BLOCK_SIZE << 1));
	int blocksPerGridLevel3 = 1 + ceil(blocksPerGridLevel2 / (BLOCK_SIZE << 1));


	// choosing the level on which to run the kernels
	// based on the size of the grids

	// if level one grid size is equal to 1 then single
	// LEVEL 1 is enough to scan the whole array
	if (blocksPerGridLevel1 == 1) {
		// record the start time
		hipEventRecord(d_start, 0);

		// execute the actual kernel
		block_scan_full<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_IN, d_OUT, len,
		NULL, 0);

		// record the stop time
		hipEventRecord(d_stop, 0);
		hipEventSynchronize(d_stop);
		hipDeviceSynchronize();
	}

	// if level two grid size is equal to 1 then two (LEVEL 2)
	// scans are required to scan the whole array
	else if (blocksPerGridLevel2 == 1) {
		// record the start time
		hipEventRecord(d_start, 0);

		// execute the actual kernels
		block_scan_full<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_IN, d_OUT, len,
				d_SUM_1, 1);
		block_scan_full<<<blocksPerGridLevel2, BLOCK_SIZE>>>(d_SUM_1,
				d_SUM_1_Scanned, blocksPerGridLevel1, NULL, 0);
		add_to_block<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_OUT, len,
				d_SUM_1_Scanned);

		// record the stop time
		hipEventRecord(d_stop, 0);
		hipEventSynchronize(d_stop);
		hipDeviceSynchronize();
	}

	// if level 3 grid size is equal to 1 then three (LEVEL 3)
	// scans are required to scan the whole array
	else if (blocksPerGridLevel3 == 1) {
		// record the start time
		hipEventRecord(d_start, 0);

		// execute the actual kernels
		block_scan_full<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_IN, d_OUT, len,
				d_SUM_1, 1);
		block_scan_full<<<blocksPerGridLevel2, BLOCK_SIZE>>>(d_SUM_1,
				d_SUM_1_Scanned, blocksPerGridLevel1, d_SUM_2, 1);
		block_scan_full<<<1, BLOCK_SIZE>>>(d_SUM_2, d_SUM_2_Scanned,
				blocksPerGridLevel2, NULL, 0);
		add_to_block<<<blocksPerGridLevel2, BLOCK_SIZE>>>(d_SUM_1_Scanned,
				blocksPerGridLevel1, d_SUM_2_Scanned);
		add_to_block<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_OUT, len,
				d_SUM_1_Scanned);

		// record the stop time
		hipEventRecord(d_stop, 0);
		hipEventSynchronize(d_stop);
		hipDeviceSynchronize();
	}

	// if none of the conditions above is met, it means that the array is too
	// large to be scanned in 3 level scan, therefore we print the error message and return
	else {
		fprintf(stderr,
				"The array size=%d is too large to be scanned with level 3 scan!\n",
				len);

		// using goto is discouraged, however, in such situations
		// where in the error conditions exit or cleanup is required
		goto cleanup;
	}

	// check whether the run was successful
	err = hipGetLastError();


	// get the duration it took for the kernels to execute
	err = hipEventElapsedTime(&d_msecs, d_start, d_stop);
	CUDA_ERROR(err, "Failed to get elapsed time");

	// print the time elapsed
	printf("Full block scan with %d elements took = %.5fmSecs\n", len, d_msecs);

	// copy the result from the device back to the host
	err = hipMemcpy(h_OUT, d_OUT, size, hipMemcpyDeviceToHost);
	CUDA_ERROR(err, "Failed to copy array OUT from device to host");

	cleanup:
	// Free device global memory
	hipFree(d_IN);
	hipFree(d_OUT);
	hipFree(d_SUM_1);
	hipFree(d_SUM_1_Scanned);
	hipFree(d_SUM_2);
	hipFree(d_SUM_2_Scanned);

	// Clean up the Device timer event objects
	hipEventDestroy(d_start);
	hipEventDestroy(d_stop);

	// Reset the device and exit
	err = hipDeviceReset();

}

/**
 * Host main routine
 */
int main(void) {

	// error code to check return clause for CUDA class
	hipError_t err = hipSuccess;

	//host time
	StopWatchInterface * timer = NULL;
	sdkCreateTimer(&timer);
	double h_msecs;

	// size of the array to add
	int numElements = 10000000;
	size_t size = numElements * sizeof(int);

	// allocate the memory on the host for the arrays
	int *h_IN = (int *) malloc(size);
	int *h_OUT = (int *) malloc(size);
	int *h_OUT_CUDA = (int *) malloc(size);

	// verify the host allocations
	if (h_IN == NULL || h_OUT == NULL || h_OUT_CUDA == NULL) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

	// initialise the host input to 1.0f
	for (int i = 0; i < numElements; i++) {
		h_IN[i] = rand() % 10;
	}

	// sequential scan
	sdkStartTimer(&timer);
	sequential_scan(h_IN, h_OUT, numElements);
	sdkStopTimer(&timer);
	h_msecs = sdkGetTimerValue(&timer);
	printf("Sequential scan on host of %d elements took = %.5fmSecs\n",
			numElements, h_msecs);

	//full block scan with bcao
	full_block_scan_BCAO(h_IN, h_OUT_CUDA, numElements);
	compare_results(h_OUT, h_OUT_CUDA, numElements);

	//full block scan without bcao
	full_block_scan(h_IN, h_OUT_CUDA, numElements);
	compare_results(h_OUT, h_OUT_CUDA, numElements);

	//simple block scan
	// create device timer
	hipEvent_t d_start, d_stop;
	float d_msecs;
	hipEventCreate(&d_start);
	hipEventCreate(&d_stop);

	// allocate memory for all the possible vectors needed for the execution
	int *d_IN = NULL;
	err = hipMalloc((void **) &d_IN, size);


	int *d_OUT = NULL;
	err = hipMalloc((void**) &d_OUT, size);


	// copy the memory from the host to the device
	err = hipMemcpy(d_IN, h_IN, size, hipMemcpyHostToDevice);


	// size of the grid for each level
	int blocksPerGridLevel1 = 1 + ((numElements - 1) / (BLOCK_SIZE * 2));

	//block scan with bcao
	// record the start time
	hipEventRecord(d_start, 0);

	// execute the actual kernel
	block_scan_full_BCAO<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_IN, d_OUT,
			numElements,
			NULL, 0);

	// record the stop time
	hipEventRecord(d_stop, 0);
	hipEventSynchronize(d_stop);
	hipDeviceSynchronize();

	// check whether the run was successful
	err = hipGetLastError();


	// get the duration it took for the kernels to execute
	err = hipEventElapsedTime(&d_msecs, d_start, d_stop);


	// print the time elapsed
	printf("Block with bank avoidance scan %d elements took = %.5fmSecs\n",
			numElements, d_msecs);

	//block scan without bcao
	// record the start time
	hipEventRecord(d_start, 0);

	// execute the actual kernel
	block_scan_full<<<blocksPerGridLevel1, BLOCK_SIZE>>>(d_IN, d_OUT,
			numElements,
			NULL, 0);

	// record the stop time
	hipEventRecord(d_stop, 0);
	hipEventSynchronize(d_stop);
	hipDeviceSynchronize();

	// check whether the run was successful
	err = hipGetLastError();


	// get the duration it took for the k// save the last element for later to improve the performance of execution
	err = hipEventElapsedTime(&d_msecs, d_start, d_stop);


	// print the time elapsed
	printf("Block scan %d elements took = %.5fmSecs\n", numElements, d_msecs);

	// Free device global memory
	hipFree(d_IN);
	hipFree(d_OUT);

	// Clean up the Device timer event objects
	hipEventDestroy(d_start);
	hipEventDestroy(d_stop);

	// Reset the device and exit
	err = hipDeviceReset();
	CUDA_ERROR(err, "Device reset error");

	// Free host memory
	free(h_IN);
	free(h_OUT);
	free(h_OUT_CUDA);

	// Clean up the Host timer
	sdkDeleteTimer(&timer);

	return 0;
}


